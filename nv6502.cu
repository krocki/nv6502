#include "hip/hip_runtime.h"
// nv6502.cu
// A GPU implementation of a 6502 CPU emulator
// Kamil M Rocki, 1/24/19

#include "nv6502.h"
#include <stdio.h>
#include <sys/time.h>
#define CHECK_ERR_CUDA(err) if (err != hipSuccess) { printf("%s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }

int read_bin(u8* mem, const char* fname) {
  FILE * file = fopen(fname, "r+");
  if (file == NULL || mem == NULL) return - 1;
  fseek(file, 0, SEEK_END);
  long int size = ftell(file);
  fclose(file);
  file = fopen(fname, "r+");
  int bytes_read = fread(mem, sizeof(u8), size, file);
  printf("read file %s, %d bytes\n", fname, bytes_read);
  return 0; fclose(file);
}

double get_time() {
  struct timeval tv; gettimeofday(&tv, NULL);
  return (tv.tv_sec + tv.tv_usec * 1e-6);
}

__device__ __host__ void print_scrn(u32 id, _6502 *n) { for (u8 i=0;i<32;i++) { for (u8 j=0;j<32;j++) { u8 v = n->mem[0x200 + j + 32*i]; if (v >= 0xa) printf("%02x ", v); else if (v > 0) printf("%2x ", v); else printf("__ "); }  printf("\n"); } }
__device__ __host__ void print_regs(u32 id, _6502 *n) { printf("[%05d] PC: %04x OP: %02x, m: %2d, d: %04x, A:%02X X:%02X Y:%02X P:%02X SP:%02X CYC:%3ld\n", id, PC, I, m, d, A, X, Y, P, SP, CY); }

#define STACK_PG 0x0100
#define ZN(x) { Z=((x)==0); S=((x)>>7) & 0x1; }
#define LDM { d=(m>2) ? r8(n,d) : d; }
#define LD_A_OR_M() u8 w=(m==1)?A:r8(n,d)
#define ST_A_OR_M() if (m!=1) w8(n,d,w); else A=w;

__device__ u8   r8    (_6502 *n, u16 a)       { return n->mem[a % MEM_SIZE];    } // byte read
__device__ void w8    (_6502 *n, u16 a, u8 v) { n->mem[a % MEM_SIZE] = v;       } // byte write
__device__ u8   f8    (_6502 *n)              { return r8(n, PC++);  } // byte fetch
//// 16-bit versions
__device__ u16  r16   (_6502 *n, u16 a)       { u16 base=a & 0xff00; return (r8(n,a) | (r8(n,base|((u8)(a+1))) << 8)); } // buggy
__device__ u16  r16_ok(_6502 *n, u16 a)       { return (r8(n,a) | (r8(n,a+1) << 8)); }
__device__ u16  f16   (_6502 *n)              { return (f8(n) | ((f8(n))<<8)); }
//
//// stack ops
__device__ u8  pop8   (_6502 *n)              { SP++; return r8(n, STACK_PG | SP);   }
__device__ u16 pop16  (_6502 *n)              { return (pop8(n) | ((pop8(n))<<8)); }
__device__ void push8 (_6502 *n, u8 v)        { w8(n, STACK_PG | SP, v); SP--; }
__device__ void push16(_6502 *n, u16 v)       { push8(n,(v>>8)); push8(n,v);  }
__device__ void jr    (_6502 *n, u8 cond)     { if (cond) { PC=(u16)d; } }
//
//// decoding addressing mode
__device__ void imp (_6502 *n) { m=0;  b=0; } // implied, 1
__device__ void acc (_6502 *n) { m=1;  b=0; } // accumulator, 1
__device__ void imm (_6502 *n) { m=2;  b=1; d=(u16)f8(n); } // immediate, 2
__device__ void zp  (_6502 *n) { m=3;  b=1; d=(u16)f8(n); } // zero page, 2
__device__ void zpx (_6502 *n) { m=4;  b=1; u8 r=f8(n); d=(r+X) & 0xff;} // zero page, x, 3
__device__ void zpy (_6502 *n) { m=5;  b=1; u8 r=f8(n); d=(r+Y) & 0xff; } // zero page, y, 3
__device__ void rel (_6502 *n) { m=6;  b=1; u8 r=f8(n); if (r<0x80) d=PC+r; else d=PC+r-0x100;} // relative, 2
__device__ void abso(_6502 *n) { m=7;  b=2; d=f16(n); } // absolute, 3
__device__ void absx(_6502 *n) { m=8;  b=2; d=f16(n); d+=X;   } // absolute, x, 3
__device__ void absy(_6502 *n) { m=9;  b=2; d=f16(n); d+=Y;  } // absolute, y, 3
__device__ void ind (_6502 *n) { m=10; b=2; d=r16(n,f16(n)); } // indirect, 3
__device__ void indx(_6502 *n) { m=11; b=1; u8 r=f8(n); d=r16(n,(u8)(r + X)); } // indirect x
__device__ void indy(_6502 *n) { m=12; b=1; u8 r=f8(n); d=r16(n,(u8)(r)); d+=Y;} // indirect y

//instructions
__device__ void _adc(_6502 *n) {
  u8 a = A; LDM; A=d+A+C; ZN(A);
  u16 t = (u16)d + (u16)a + (u16)C; C=(t > 0xff);
  V = (!((a^d) & 0x80)) && (((a^A) & 0x80)>0 );
} //   Add Memory to Accumulator with Carry

__device__ void _sbc(_6502 *n) {
  u8 a = A; LDM; A=A-d-(1-C); ZN(A);
  s16 t = (s16)a - (s16)d - (1-(s16)C); C=(t >= 0x0);
  V = (((a^d) & 0x80)>0) && (((a^A) & 0x80)>0);
} //   Subtract Memory from Accumulator with Borrow

__device__ void _cp (_6502 *n, u8 _a, u8 _b) { u8 r=_a-_b; C=(_a>=_b); ZN(r); }
__device__ void _ora(_6502 *n) { LDM; A|=d; ZN(A); } //   "OR" Memory with Accumulator
__device__ void _and(_6502 *n) { LDM; A&=d; ZN(A); } //   "AND" Memory with Accumulator
__device__ void _eor(_6502 *n) { LDM; A^=d; ZN(A); } //   "XOR" Memory with Accumulator
__device__ void _cmp(_6502 *n) { LDM; _cp(n,A,d); } //   Compare Memory and Accumulator
__device__ void _cpx(_6502 *n) { LDM; _cp(n,X,d); } //   Compare Memory and Index X
__device__ void _cpy(_6502 *n) { LDM; _cp(n,Y,d); } //   Compare Memory and Index Y
__device__ void _bcc(_6502 *n) { jr(n,!C); } //   Branch on Carry Clear
__device__ void _bcs(_6502 *n) { jr(n,C);  } //   Branch on Carry Set
__device__ void _beq(_6502 *n) { jr(n,Z);  } //   Branch on Result Zero
__device__ void _bit(_6502 *n) { LDM; S=(d>>7) & 1; V=(d>>6) & 1; Z=(d & A)==0; } // Test Bits in Memory with A
__device__ void _bmi(_6502 *n) { jr(n, S);  } //  Branch on Result Minus
__device__ void _bne(_6502 *n) { jr(n,!Z); } //   Branch on Result not Zero
__device__ void _bpl(_6502 *n) { jr(n,!S); } //   Branch on Result Plus
__device__ void _brk(_6502 *n) { B=1;    } //   Force Break
__device__ void _bvc(_6502 *n) { jr(n,!V); } //   Branch on Overflow Clear
__device__ void _bvs(_6502 *n) { jr(n, V);  } //   Branch on Overflow Set
__device__ void _clc(_6502 *n) { C=0; } //   Clear Carry Flag
__device__ void _cld(_6502 *n) { D=0; } //   Clear Decimal Mode
__device__ void _cli(_6502 *n) { I=0; } //   Clear interrupt Disable Bit
__device__ void _clv(_6502 *n) { V=0; } //   Clear Overflow Flag
__device__ void _dec(_6502 *n) { u16 d0 = d; LDM; d--; d &= 0xff; ZN(d); w8(n,d0,d); } //   Decrement Memory by One
__device__ void _dex(_6502 *n) { X--; ZN(X); } //   Decrement Index X by One
__device__ void _dey(_6502 *n) { Y--; ZN(Y); } //   Decrement Index Y by One
__device__ void _inc(_6502 *n) { u16 d0=d; LDM; d++; d &= 0xff; ZN(d); w8(n,d0,d); d=d0; } // Incr Memory by One
__device__ void _inx(_6502 *n) { X++; ZN(X); } //   Increment Index X by One
__device__ void _iny(_6502 *n) { Y++; ZN(Y); } //   Increment Index Y by One
__device__ void _jmp(_6502 *n) { PC=d;} //   Jump to New Location
__device__ void _jsr(_6502 *n) { push16(n,PC-1); PC=d; } //   Jump to New Location Saving Return Address
__device__ void _lda(_6502 *n) { LDM; A=d; ZN(A); } //   Load Accumulator with Memory
__device__ void _ldx(_6502 *n) { LDM; X=d; ZN(X); } //   Load Index X with Memory
__device__ void _ldy(_6502 *n) { LDM; Y=d; ZN(Y); } //   Load Index Y with Memory
__device__ void _lsr(_6502 *n) { LD_A_OR_M(); C=w & 1; w>>=1; ZN(w); ST_A_OR_M(); } // Shift Right One Bit
__device__ void _asl(_6502 *n) { LD_A_OR_M(); C=(w>>7) & 1; w<<=1; ZN(w); ST_A_OR_M();} // Shift Left One Bit
__device__ void _rol(_6502 *n) { LD_A_OR_M(); u8 c = C; C=(w>>7) & 1; w=(w<<1) | c; ZN(w); ST_A_OR_M(); } // Rotate One Bit Left (Memory or Accumulator)
__device__ void _ror(_6502 *n) { LD_A_OR_M(); u8 c = C; C=(w & 1); w=(w>>1) | (c<<7); ZN(w); ST_A_OR_M(); } //   Rotate One Bit Right (Memory or Accumulator)
__device__ void _nop(_6502 *n) { /* No Operation */ }
__device__ void _pha(_6502 *n) { push8(n, A); } //   Push Accumulator on Stack
__device__ void _php(_6502 *n) { push8(n, P | 0x10); } //   Push Processor Status on Stack
__device__ void _pla(_6502 *n) { A=pop8(n); Z=(A==0); S=(A>>7)&0x1;} //   Pull Accumulator from Stack
__device__ void _plp(_6502 *n) { P=pop8(n) & 0xef | 0x20;  } //   Pull Processor Status from Stack
__device__ void _rti(_6502 *n) { P=(pop8(n) & 0xef) | 0x20; PC=pop16(n); } //   Return from Interrupt
__device__ void _rts(_6502 *n) { PC=pop16(n)+1;} //   Return from Subroutine
__device__ void _sec(_6502 *n) { C=1;} //   Set Carry Flag
__device__ void _sed(_6502 *n) { D=1;} //   Set Decimal Mode
__device__ void _sei(_6502 *n) { I=1;} //   Set Interrupt Disable Status
__device__ void _sta(_6502 *n) { w8(n,d,A);} //   Store Accumulator in Memory
__device__ void _stx(_6502 *n) { w8(n,d,X);} //   Store Index X in Memory
__device__ void _sty(_6502 *n) { w8(n,d,Y);} //   Store Index Y in Memory
__device__ void _tax(_6502 *n) { X=A; ZN(X); } //   Transfer Accumulator to Index X
__device__ void _tay(_6502 *n) { Y=A; ZN(Y); } //   Transfer Accumulator to Index Y
__device__ void _tsx(_6502 *n) { X=SP;ZN(X); } //   Transfer Stack Pointer to Index X
__device__ void _txa(_6502 *n) { A=X; ZN(A); } //   Transfer Index X to Accumulator
__device__ void _txs(_6502 *n) { SP=X; } //   Transfer Index X to Stack Pointer
__device__ void _tya(_6502 *n) { A=Y; ZN(A); } //   Transfer Index Y to Accumulator
// undocumented
__device__ void _lax(_6502 *n) { _lda(n); X=A; ZN(A); } // lda, ldx
__device__ void _sax(_6502 *n) { w8(n,d,A&X); }
__device__ void _dcp(_6502 *n) { _dec(n); _cp(n,A,d); }
__device__ void _isb(_6502 *n) { _inc(n); _sbc(n); }
__device__ void _slo(_6502 *n) { _asl(n); _ora(n); }
__device__ void _rla(_6502 *n) { _rol(n); _and(n); }
__device__ void _sre(_6502 *n) { _lsr(n); _eor(n); }
__device__ void _rra(_6502 *n) { _ror(n); _adc(n); }

__device__ void *addrtable[256] = {
  &imp, &indx,&imp,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &acc,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpx,&zpx,&imp,&absy,&imp,&absy,&absx,&absx,&absx,&absx,
  &abso,&indx,&imp,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &acc,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpx,&zpx,&imp,&absy,&imp,&absy,&absx,&absx,&absx,&absx,
  &imp, &indx,&imp,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &acc,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpx,&zpx,&imp,&absy,&imp,&absy,&absx,&absx,&absx,&absx,
  &imp, &indx,&imp,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &acc,&imm, &ind, &abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpx,&zpx,&imp,&absy,&imp,&absy,&absx,&absx,&absx,&absx,
  &imm, &indx,&imm,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &imp,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpy,&zpy,&imp,&absy,&imp,&absy,&absx,&absx,&absy,&absy,
  &imm, &indx,&imm,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &imp,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpy,&zpy,&imp,&absy,&imp,&absy,&absx,&absx,&absy,&absy,
  &imm, &indx,&imm,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &imp,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpx,&zpx,&imp,&absy,&imp,&absy,&absx,&absx,&absx,&absx,
  &imm, &indx,&imm,&indx,&zp, &zp, &zp, &zp, &imp,&imm, &imp,&imm, &abso,&abso,&abso,&abso,
  &rel, &indy,&imp,&indy,&zpx,&zpx,&zpx,&zpx,&imp,&absy,&imp,&absy,&absx,&absx,&absx,&absx};

__device__ void *optable[256] = { // opcode -> functions map
  &_brk,&_ora,&_nop,&_slo,&_nop,&_ora,&_asl,&_slo,&_php,&_ora,&_asl,&_nop,&_nop,&_ora,&_asl,&_slo,
  &_bpl,&_ora,&_nop,&_slo,&_nop,&_ora,&_asl,&_slo,&_clc,&_ora,&_nop,&_slo,&_nop,&_ora,&_asl,&_slo,
  &_jsr,&_and,&_nop,&_rla,&_bit,&_and,&_rol,&_rla,&_plp,&_and,&_rol,&_nop,&_bit,&_and,&_rol,&_rla,
  &_bmi,&_and,&_nop,&_rla,&_nop,&_and,&_rol,&_rla,&_sec,&_and,&_nop,&_rla,&_nop,&_and,&_rol,&_rla,
  &_rti,&_eor,&_nop,&_sre,&_nop,&_eor,&_lsr,&_sre,&_pha,&_eor,&_lsr,&_nop,&_jmp,&_eor,&_lsr,&_sre,
  &_bvc,&_eor,&_nop,&_sre,&_nop,&_eor,&_lsr,&_sre,&_cli,&_eor,&_nop,&_sre,&_nop,&_eor,&_lsr,&_sre,
  &_rts,&_adc,&_nop,&_rra,&_nop,&_adc,&_ror,&_rra,&_pla,&_adc,&_ror,&_nop,&_jmp,&_adc,&_ror,&_rra,
  &_bvs,&_adc,&_nop,&_rra,&_nop,&_adc,&_ror,&_rra,&_sei,&_adc,&_nop,&_rra,&_nop,&_adc,&_ror,&_rra,
  &_nop,&_sta,&_nop,&_sax,&_sty,&_sta,&_stx,&_sax,&_dey,&_nop,&_txa,&_nop,&_sty,&_sta,&_stx,&_sax,
  &_bcc,&_sta,&_nop,&_nop,&_sty,&_sta,&_stx,&_sax,&_tya,&_sta,&_txs,&_nop,&_nop,&_sta,&_nop,&_nop,
  &_ldy,&_lda,&_ldx,&_lax,&_ldy,&_lda,&_ldx,&_lax,&_tay,&_lda,&_tax,&_nop,&_ldy,&_lda,&_ldx,&_lax,
  &_bcs,&_lda,&_nop,&_lax,&_ldy,&_lda,&_ldx,&_lax,&_clv,&_lda,&_tsx,&_lax,&_ldy,&_lda,&_ldx,&_lax,
  &_cpy,&_cmp,&_nop,&_dcp,&_cpy,&_cmp,&_dec,&_dcp,&_iny,&_cmp,&_dex,&_nop,&_cpy,&_cmp,&_dec,&_dcp,
  &_bne,&_cmp,&_nop,&_dcp,&_nop,&_cmp,&_dec,&_dcp,&_cld,&_cmp,&_nop,&_dcp,&_nop,&_cmp,&_dec,&_dcp,
  &_cpx,&_sbc,&_nop,&_isb,&_cpx,&_sbc,&_inc,&_isb,&_inx,&_sbc,&_nop,&_sbc,&_cpx,&_sbc,&_inc,&_isb,
  &_beq,&_sbc,&_nop,&_isb,&_nop,&_sbc,&_inc,&_isb,&_sed,&_sbc,&_nop,&_isb,&_nop,&_sbc,&_inc,&_isb
};

// use local mem? this is usually faster
#define LMEM

__global__ void step(_6502* states, int steps, int num_threads) {
  int i = blockDim.x * blockIdx.x + threadIdx.x; // thread idx
  if (i < num_threads) {
#ifdef LMEM // use local memory
    _6502 ln = states[i]; _6502 *n = &ln;
#else       // operate directly on global mem
    _6502 *n = &states[i];
#endif

    for (int j = 0; j < steps; ++j) {
      u8 op = f8(n); I = op;               // fetch next byte
      ((void(*)(_6502*))addrtable[op])(n); // decode addr mode
      ((void(*)(_6502*))  optable[op])(n); // execute
      CY++;                                // increment cycle count
    }

#ifdef LMEM // update from local mem
    states[i] = ln;
#endif
  }
}

void reset(_6502 *n, u16 _PC, u8 _SP) {
  PC=_PC; A=0x00; X=0x00; P=0x24; SP=_SP; CY=0; memset(n->mem, '\0', MEM_SIZE);
}

// wrapper for CUDA call
void gpu_step(_6502* states, u32 steps, u32 num_blocks, u32 threads_per_block) {
  step<<<num_blocks, threads_per_block>>>(states, steps, num_blocks * threads_per_block);
}

hipError_t err = hipSuccess; // for checking CUDA errors
_6502* d_regs = NULL;
_6502    *h_in_regs;
_6502    *h_out_regs;
int num_blocks, threads_per_block, iters, steps, num_threads;

int init(int blks, int threads, int _iters, int _steps) {

  num_blocks = blks; threads_per_block = threads;  iters = _iters; steps = _steps;
  num_threads = num_blocks * threads_per_block;

  // allocate _6502 registers / state
  h_in_regs   = (_6502 *) malloc(num_threads * sizeof(_6502));
  h_out_regs  = (_6502 *) malloc(num_threads * sizeof(_6502));

  printf("  main: allocating %zu device bytes\n", num_threads * sizeof(_6502));
  err = hipMalloc((void **)&d_regs, num_threads * sizeof(_6502) ); CHECK_ERR_CUDA(err);
  return 0;
}

int teardown() {

  err = hipFree(d_regs); CHECK_ERR_CUDA(err);
  free(h_in_regs); free(h_out_regs);
  return 0;
}

int run(char* file) {
  printf("  main: running %d blocks * %d threads (%d threads total)\n", num_blocks, threads_per_block, num_threads);
  // resetting all instances
  for (u32 i = 0; i < num_threads; i++) {
    reset(&h_in_regs[i], 0x0600, 0xfe); read_bin(&h_in_regs[i].mem[0x0600],file);
  }

  printf("  main: copying host -> device\n");
  err = hipMemcpy(d_regs, h_in_regs, sizeof(_6502 ) * num_threads, hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);

  for (int j = 0; j < iters; j++ ) {

    double start_time = get_time();
    hipDeviceSynchronize();
    ///
    gpu_step(d_regs, steps, num_blocks, threads_per_block);
    ///
    hipDeviceSynchronize();
    double walltime = get_time() - start_time;
    err = hipGetLastError(); CHECK_ERR_CUDA(err);
    printf("  main: kernel time = %.6f s, %2.6f us/step, %5.3f MHz\n", walltime,
        1e6 * (walltime/(steps * num_threads)), ((steps * num_threads)/walltime)/1e6);

    printf("  main: copying device -> host\n");
    err = hipMemcpy(h_out_regs, d_regs, sizeof(_6502) * num_threads, hipMemcpyDeviceToHost); CHECK_ERR_CUDA(err);

    for (u32 i = 0; i < num_threads; i++) { print_regs(i, &h_out_regs[i]); print_scrn(i, &h_out_regs[i]); }
  }

  return 0;
}

int main(int argc, void** argv) {
  init(1,1,32,1024);
  run("sierp.bin");
  teardown();
}

